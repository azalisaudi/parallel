#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Macro for checking CUDA errors
#define CUDA_CHECK(call)                                                          \
    do {                                                                          \
        hipError_t err = call;                                                   \
        if (err != hipSuccess) {                                                 \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                          \
                    __FILE__, __LINE__, hipGetErrorString(err));                 \
            exit(EXIT_FAILURE);                                                   \
        }                                                                         \
    } while (0)

// --- KERNEL CODE (Device) ---
// This kernel performs the parallel reduction within a single block.
__global__ void reduceSum(int *g_input, int *g_output) {
    // Dynamic shared memory allocation
    extern __shared__ int s_data[]; 
    
    // Global index: determines the element in the input array this thread reads
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Local index: the thread's position within its block (0 to blockDim.x - 1)
    unsigned int tid = threadIdx.x; 
    
    // Load data from global memory to shared memory
    s_data[tid] = g_input[i];
    
    // Synchronize to ensure all data is loaded before starting the reduction
    __syncthreads();
    
    // Logarithmic reduction (tree reduction)
    for (unsigned int s = blockDim.x / 2; s > 0; s /= 2) {
        if (tid < s) {
            s_data[tid] += s_data[tid + s];
        }
        // Synchronize after each step to ensure all additions are complete
        __syncthreads();
    }
    
    // The final sum for the block is in s_data[0]. 
    // Thread 0 writes the result to global memory.
    if (tid == 0) {
        g_output[blockIdx.x] = s_data[0];
    }
}

// --- HOST CODE (CPU) ---
int main() {
    const int N = 1024 * 1024; // Total number of elements
    const int BLOCK_SIZE = 512;  // Threads per block
    
    // Calculate Grid Size
    // Each block will process BLOCK_SIZE elements and produce one partial sum.
    const int NUM_BLOCKS = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // Size in bytes
    const size_t arraySize = N * sizeof(int);
    const size_t outputSize = NUM_BLOCKS * sizeof(int);

    // Host memory allocation
    int *h_input = (int*)malloc(arraySize);
    
    // Initialize host input data (e.g., set all elements to 1)
    long long expectedSum = 0;
    for (int i = 0; i < N; i++) {
        h_input[i] = 1;
        expectedSum += h_input[i];
    }
    
    // Device memory pointers
    int *d_input, *d_output;

    // 1. Memory Allocation on Device
    CUDA_CHECK(hipMalloc((void**)&d_input, arraySize));
    CUDA_CHECK(hipMalloc((void**)&d_output, outputSize));
    
    // 2. Data Transfer: Host to Device
    CUDA_CHECK(hipMemcpy(d_input, h_input, arraySize, hipMemcpyHostToDevice));

    // 3. Kernel Launch (First Pass: Partial Sums)
    // The shared memory size is BLOCK_SIZE * sizeof(int)
    reduceSum<<<NUM_BLOCKS, BLOCK_SIZE, BLOCK_SIZE * sizeof(int)>>>(d_input, d_output);
    
    // Synchronize to wait for all blocks to finish their partial sums
    CUDA_CHECK(hipDeviceSynchronize());
    
    // 4. Data Transfer: Device to Host (Partial Sums)
    // We reuse h_input for simplicity to store the partial sums
    int *h_output = (int*)malloc(outputSize);
    CUDA_CHECK(hipMemcpy(h_output, d_output, outputSize, hipMemcpyDeviceToHost));
    
    // --- FINAL REDUCTION ON CPU (Second Pass) ---
    // For simplicity, we sum the partial results on the CPU.
    // For larger problems, a second, smaller kernel is often used.
    long long finalSum = 0;
    for (int i = 0; i < NUM_BLOCKS; i++) {
        finalSum += h_output[i];
    }

    // 5. Verification and Cleanup
    printf("Array Size (N): %d\n", N);
    printf("Blocks Launched: %d\n", NUM_BLOCKS);
    printf("Expected Sum: %lld\n", expectedSum);
    printf("CUDA Final Sum: %lld\n", finalSum);

    if (finalSum == expectedSum) {
        printf("Result: SUCCESS! 🎉\n");
    } else {
        printf("Result: FAILURE! 😔\n");
    }

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
